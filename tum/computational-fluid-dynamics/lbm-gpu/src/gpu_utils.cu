#include "gpu_utils.h"


int hasCudaGpu(){
	int devices = 0;
	hipError_t err = hipGetDeviceCount(&devices);
	return (devices > 0 && err == hipSuccess) ? 1 : 0;
}
