#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "lbm_definitions.h"
#include "collision_gpu.h"

//restricts 3D blocks to have 512 threads (max 1024)
#define BLOCK_SIZE 8

#define cudaErrorCheck(ans){ cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, char *file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr,"CUDA Assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__constant__ double tau_d, xlength_d;
//TODO:Move lattice constants to constant memory
//__constant__ int LATTICE_VELOCITIES_D[Q_LBM];
//LATTICE_WEIGHTS_D[D_LBM][Q_LBM]

__device__ static const int LATTICE_VELOCITIES_D[19][3] = {
    {0,-1,-1},{-1,0,-1},{0,0,-1},{1,0,-1},{0,1,-1},{-1,-1,0},{0,-1,0},{1,-1,0},
    {-1,0,0}, {0,0,0},  {1,0,0}, {-1,1,0},{0,1,0}, {1,1,0},  {0,-1,1},{-1,0,1},
    {0,0,1},  {1,0,1},  {0,1,1}
};
__device__ static const double LATTICE_WEIGHTS_D[19] = {
    1.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0,
    2.0/36.0, 12.0/36.0,2.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0,
    2.0/36.0, 1.0/36.0, 1.0/36.0
};


/** computes the density from the particle distribution functions stored at currentCell.
 *  currentCell thus denotes the address of the first particle distribution function of the
 *  respective cell. The result is stored in density.
 */
__device__ void ComputeDensity(double *current_cell, double *density, int *error){
    int i; *density=0;
    for(i=0;i<Q_LBM;i++)
        *density+=current_cell[i];

    /* Density should be close to a unit (ρ~1) */
    if((*density-1.0)>EPS)
    	error[0]++;
//        ERROR("Density dropped below error tolerance.");
}


/** computes the velocity within currentCell and stores the result in velocity */
__device__ void ComputeVelocity(double *current_cell, double *density, double *velocity){
    int i;
    /* NOTE:Indeces are hardcoded because of the possible performance gains and since
     * we do not have alternating D */
    velocity[0]=0;
    velocity[1]=0;
    velocity[2]=0;

    for(i=0;i<Q_LBM;i++){
        velocity[0]+=current_cell[i]*LATTICE_VELOCITIES_D[i][0];
        velocity[1]+=current_cell[i]*LATTICE_VELOCITIES_D[i][1];
        velocity[2]+=current_cell[i]*LATTICE_VELOCITIES_D[i][2];
    }

    velocity[0]/=*density;
    velocity[1]/=*density;
    velocity[2]/=*density;
}


/** computes the equilibrium distributions for all particle distribution functions of one
 *  cell from density and velocity and stores the results in feq.
 */
__device__ void ComputeFeq(double *density, double * velocity, double *feq, int *error){
    int i;
    double s1, s2, s3; /* summands */
    for(i=0;i<Q_LBM;i++){
        s1 = LATTICE_VELOCITIES_D[i][0]*velocity[0]+LATTICE_VELOCITIES_D[i][1]*velocity[1]+
        		LATTICE_VELOCITIES_D[i][2]*velocity[2];
        s2 = s1*s1;
        s3 = velocity[0]*velocity[0]+velocity[1]*velocity[1]+velocity[2]*velocity[2];

        feq[i]=LATTICE_WEIGHTS_D[i]*(*density)*(1+s1*C_S_POW2_INV+s2*C_S_POW4_INV/2.0-s3*C_S_POW2_INV/2.0);

        /* Probability distribution function can not be less than 0 */
        if (feq[i] < 0)
        	error[1]++;
//            ERROR("Probability distribution function can not be negative.");
    }
}


/** Computes the post-collision distribution functions according to the BGK update rule and
 *  stores the results again at the same position.
 */
__device__ void ComputePostCollisionDistributions(double *current_cell, double *feq, int *error){
    int i;
    for(i=0;i<Q_LBM;i++){
        current_cell[i]=current_cell[i]-(current_cell[i]-feq[i])/tau_d;

        /* Probability distribution function can not be less than 0 */
        if (current_cell[i] < 0)
        	error[2]++;
//            ERROR("Probability distribution function can not be negative.");
    }
}


__global__ void DoColision(double *collide_field_d, int *error_d){
	//	__syncthreads(); to use after reading data into shared memory
	int x = 1+threadIdx.x+blockIdx.x*blockDim.x;
	int y = 1+threadIdx.y+blockIdx.y*blockDim.y;
	int z = 1+threadIdx.z+blockIdx.z*blockDim.z;
	double density, velocity[3], feq[Q_LBM], *currentCell;
	int step=xlength_d+2;

	//check that indices are within the bounds since there could be more threads than needed
	if (0<x && x<(step-1) && 0<y && y<(step-1) && 0<z && z<(step-1) && !error_d[0] && !error_d[1] && !error_d[2]){
		currentCell=&collide_field_d[Q_LBM*(x+y*step+z*step*step)];
		ComputeDensity(currentCell,&density,error_d);
		ComputeVelocity(currentCell,&density,velocity);
		ComputeFeq(&density,velocity,feq,error_d);
		ComputePostCollisionDistributions(currentCell,feq,error_d);
	}
}


void DoCollisionCuda(double *collide_field, int *flag_field, double tau, int xlength){
	double *collide_field_d=NULL;
	int num_cells = pow(xlength+2, D_LBM), *error_d, *error;
	size_t size = Q_LBM*num_cells*sizeof(double);

	error = (int*)malloc(3*sizeof(int));
	error[0]=0;error[1]=0;error[2]=0;

	//initialize working data
	cudaErrorCheck(hipMalloc(&collide_field_d, size));
	cudaErrorCheck(hipMemcpy(collide_field_d, collide_field, size, hipMemcpyHostToDevice));

	cudaErrorCheck(hipMalloc(&error_d, 3*sizeof(int)));
	cudaErrorCheck(hipMemcpy(error_d, error, 3*sizeof(int), hipMemcpyHostToDevice));
	//initialize constant data
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), &tau, sizeof(double), 0, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice));
//	hipMemcpyToSymbol(HIP_SYMBOL(LATTICE_VELOCITIES_D), &LATTICE_VELOCITIES, D_LBM*Q_LBM*sizeof(int), 0, hipMemcpyHostToDevice);
//	hipMemcpyToSymbol(HIP_SYMBOL(LATTICE_WEIGHTS_D), &LATTICE_WEIGHTS, Q_LBM*sizeof(double), 0, hipMemcpyHostToDevice);

	//define grid structure
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+block.x-1)/block.x, (xlength+block.y-1)/block.y, (xlength+block.z-1)/block.z);

	//perform collision
//	printf("\n Performing collision with grid(%d,%d,%d) and block(%d,%d,%d) \n", grid.x,grid.y,grid.z,
//			block.x,block.y,block.z);
	DoColision<<<grid, block>>>(collide_field_d, error_d);

	cudaErrorCheck(hipPeekAtLastError());
	//	hipDeviceSynchronize(); this is already in memcpy
	//copy data back to host
	cudaErrorCheck(hipMemcpy(collide_field, collide_field_d, size, hipMemcpyDeviceToHost));
	cudaErrorCheck(hipMemcpy(error, error_d, 3*sizeof(int), hipMemcpyDeviceToHost));
	printf("Density: %d Feq: %d Postcol: %d\n", error[0], error[1], error[2]);

	//free device memory
	cudaErrorCheck(hipFree(collide_field_d));
	cudaErrorCheck(hipFree(error_d));

	free(error);
}
