#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "lbm_definitions.h"
#include "collision_gpu.h"

//restricts 3D blocks to have 512 threads (limits: 512 CC<2.x; 1024 CC>2.x)
#define BLOCK_SIZE 8

/**
 * Checks the returned hipError_t and prints corresponding message in case of error.
 */
#define cudaErrorCheck(ans){ cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, char *file, int line, bool abort=true){
	if (code != hipSuccess){
		fprintf(stderr,"CUDA Assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__constant__ double tau_d;
__constant__ int xlength_d;
//TODO:move lattice constants to gpu constant memory
__device__ static const int LATTICE_VELOCITIES_D[19][3] = {
    {0,-1,-1},{-1,0,-1},{0,0,-1},{1,0,-1},{0,1,-1},{-1,-1,0},{0,-1,0},{1,-1,0},
    {-1,0,0}, {0,0,0},  {1,0,0}, {-1,1,0},{0,1,0}, {1,1,0},  {0,-1,1},{-1,0,1},
    {0,0,1},  {1,0,1},  {0,1,1}
};
__device__ static const double LATTICE_WEIGHTS_D[19] = {
    1.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0,
    2.0/36.0, 12.0/36.0,2.0/36.0, 1.0/36.0, 2.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0,
    2.0/36.0, 1.0/36.0, 1.0/36.0
};


/**
 * Computes the density from the particle distribution functions stored at currentCell.
 * currentCell thus denotes the address of the first particle distribution function of the
 * respective cell. The result is stored in density.
 */
__device__ void ComputeDensity(double *current_cell, double *density){
    int i; *density=0;
    for(i=0;i<Q_LBM;i++)
        *density+=current_cell[i];

    /* Density should be close to a unit (ρ~1) */
//    if((*density-1.0)>EPS)
//        ERROR("Density dropped below error tolerance.");
}


/**
 * Computes the velocity within currentCell and stores the result in velocity
 */
__device__ void ComputeVelocity(double *current_cell, double *density, double *velocity){
    int i;
    velocity[0]=0;
    velocity[1]=0;
    velocity[2]=0;

    for(i=0;i<Q_LBM;i++){
        velocity[0]+=current_cell[i]*LATTICE_VELOCITIES_D[i][0];
        velocity[1]+=current_cell[i]*LATTICE_VELOCITIES_D[i][1];
        velocity[2]+=current_cell[i]*LATTICE_VELOCITIES_D[i][2];
    }

    velocity[0]/=*density;
    velocity[1]/=*density;
    velocity[2]/=*density;
}


/**
 * Computes the equilibrium distributions for all particle distribution functions of one
 * cell from density and velocity and stores the results in feq.
 */
__device__ void ComputeFeq(double *density, double *velocity, double *feq){
    int i;
    double s1, s2, s3;
    for(i=0;i<Q_LBM;i++){
        s1 = LATTICE_VELOCITIES_D[i][0]*velocity[0]+LATTICE_VELOCITIES_D[i][1]*velocity[1]+
        		LATTICE_VELOCITIES_D[i][2]*velocity[2];
        s2 = s1*s1;
        s3 = velocity[0]*velocity[0]+velocity[1]*velocity[1]+velocity[2]*velocity[2];

        feq[i]=LATTICE_WEIGHTS_D[i]*(*density)*(1+s1*C_S_POW2_INV+s2*C_S_POW4_INV/2.0-s3*C_S_POW2_INV/2.0);

        /* Probability distribution function can not be less than 0 */
//        if (feq[i] < 0)
//            ERROR("Probability distribution function can not be negative.");
    }
}


/**
 * Computes the post-collision distribution functions according to the BGK update rule and
 * stores the results again at the same position.
 */
__device__ void ComputePostCollisionDistributions(double *current_cell, double *feq){
    int i;
    for(i=0;i<Q_LBM;i++){
        current_cell[i]=current_cell[i]-(current_cell[i]-feq[i])/tau_d;

        /* Probability distribution function can not be less than 0 */
//        if (current_cell[i] < 0)
//            ERROR("Probability distribution function can not be negative.");
    }
}

/**
 * Performs the actual collision computation
 */
__global__ void DoColision(double *collide_field_d){
	//	__syncthreads(); to use after reading data into shared memory
	double density, velocity[D_LBM], feq[Q_LBM], *currentCell;
	int x = 1+threadIdx.x+blockIdx.x*blockDim.x;
	int y = 1+threadIdx.y+blockIdx.y*blockDim.y;
	int z = 1+threadIdx.z+blockIdx.z*blockDim.z;
	int step = xlength_d+2;
	int idx = x+y*step+z*step*step;

	//check that indices are within the bounds since there could be more threads than needed
	if (x<(step-1) && y<(step-1) && z<(step-1)){
		currentCell=&collide_field_d[Q_LBM*idx];
		ComputeDensity(currentCell,&density);
		ComputeVelocity(currentCell,&density,velocity);
		ComputeFeq(&density,velocity,feq);
		ComputePostCollisionDistributions(currentCell,feq);
	}
}


void DoCollisionCuda(double *collide_field, int *flag_field, double tau, int xlength){
	double *collide_field_d=NULL;
	int num_cells = pow(xlength+2, D_LBM);
	size_t collide_field_size = Q_LBM*num_cells*sizeof(double);

	//initialize working data
	cudaErrorCheck(hipMalloc(&collide_field_d, collide_field_size));
	cudaErrorCheck(hipMemcpy(collide_field_d, collide_field, collide_field_size, hipMemcpyHostToDevice));

	//initialize constant data
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), &tau, sizeof(double), 0, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice));

	//define grid structure
	//NOTE:redundant threads for boundary cells are not accounted for
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+block.x-1)/block.x, (xlength+block.y-1)/block.y, (xlength+block.z-1)/block.z);

	//perform collision
	DoColision<<<grid,block>>>(collide_field_d);
	cudaErrorCheck(hipPeekAtLastError());

	//copy data back to host
	cudaErrorCheck(hipMemcpy(collide_field, collide_field_d, collide_field_size, hipMemcpyDeviceToHost));

	//free device memory
	cudaErrorCheck(hipFree(collide_field_d));
}


/**
 * Performs the actual streaming computation
 */
__global__ void DoStreaming(double *stream_field_d, double *collide_field_d){
	//	__syncthreads(); to use after reading data into shared memory
	int x = 1+threadIdx.x+blockIdx.x*blockDim.x;
	int y = 1+threadIdx.y+blockIdx.y*blockDim.y;
	int z = 1+threadIdx.z+blockIdx.z*blockDim.z;
	int step = xlength_d+2, idx = x+y*step+z*step*step, nx, ny, nz, i;

	//check that indices are within the bounds since there could be more threads than needed
	if (x<(step-1) && y<(step-1) && z<(step-1)){
		for(i=0;i<Q_LBM;i++){
			nx=x-LATTICE_VELOCITIES_D[i][0];
			ny=y-LATTICE_VELOCITIES_D[i][1];
			nz=z-LATTICE_VELOCITIES_D[i][2];

			stream_field_d[Q_LBM*idx+i]=
					collide_field_d[Q_LBM*(nx+ny*step+nz*step*step)+i];
		}
	}
}


void DoStreamingCuda(double *collide_field, double *stream_field, int *flag_field, int xlength){
	double *collide_field_d=NULL, *stream_field_d=NULL;
	int num_cells = pow(xlength+2, D_LBM);
	size_t computational_field_size = Q_LBM*num_cells*sizeof(double);

	//initialize working data
	cudaErrorCheck(hipMalloc(&collide_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(collide_field_d, collide_field, computational_field_size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMalloc(&stream_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(stream_field_d, stream_field, computational_field_size, hipMemcpyHostToDevice));

	//initialize constant data
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice));

	//define grid structure
	//NOTE:redundant threads for boundary cells are not accounted for
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+block.x-1)/block.x, (xlength+block.y-1)/block.y, (xlength+block.z-1)/block.z);

	//perform streaming
	DoStreaming<<<grid,block>>>(stream_field_d, collide_field_d);
	cudaErrorCheck(hipPeekAtLastError());

	//copy data back to host
	cudaErrorCheck(hipMemcpy(stream_field, stream_field_d, computational_field_size, hipMemcpyDeviceToHost));

	//free device memory
	cudaErrorCheck(hipFree(collide_field_d));
	cudaErrorCheck(hipFree(stream_field_d));
}
