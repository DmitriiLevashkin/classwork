#include "hip/hip_runtime.h"
#include "collision_gpu.h"

__global__
void DoNothing(){
//    int i = blockDim.x*blockIdx.x + threadIdx.x;
//    cuPrintf("Hi there pretty! Wanna come up for a tea? My address is: block #%d thread #%d gridwise #%d",
//    		blockIdx.x, threadIdx.x, i);
}

void CudaTest(double *collide_field, size_t size){
	double *collide_field_d=NULL;

	//Ninja code
	hipMalloc(&collide_field_d, size);
	hipMemcpy(collide_field_d, collide_field, size, hipMemcpyHostToDevice);
	DoNothing<<<1, 6>>>();
	hipMemcpy(collide_field, collide_field_d, size, hipMemcpyDeviceToHost);
	hipFree(collide_field_d);
}
